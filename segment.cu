#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<opencv2\opencv.hpp>
#include<>
#include"utils.h"
using namespace std;
using namespace cv;
static __device__ double compute_distance(const Component &center, const Pixel &point, const double &area, int sign)
{
	double colorDiff = (center.lMean - point.l) * (center.lMean - point.l) + (center.aMean - point.a) * (center.aMean - point.a) + (center.bMean - point.b) * (center.bMean - point.b);
	double spaceDiff = (center.xMean - point.x) * (center.xMean - point.x) + (center.yMean - point.y) * (center.yMean - point.y);
	double diff = 0.f;
	if (sign == 0)
		diff = colorDiff;
	else if (sign == 1)
	{

		diff = colorDiff + spaceDiff * 10;
	}
	return diff;
}

__device__ int _max(int x, int y)
{
	return x > y ? x : y;
}

__device__ int _min(int x, int y)
{
	return x < y ? x : y;
}

extern __global__ void clustering(const int* img,int* imgLabels, Component* _components, const Score* scores, const bool* is_test_, const int* _height_, const int* _width_, int* xbuff_, int* ybuff_)
{
	int i = threadIdx.x;
	
	Component &component = _components[scores[i].label];
	if (component.score < 1.0)
		return;
	int count = 1;
	const int numDirections = 4;
	const bool is_test = *is_test_;
	const int _width = *_width_;
	const int _height = *_height_;
	const int sz = _height * _width;
	const int dxs[numDirections] = { -1, 0, 1, 0 };
	const int dys[numDirections] = { 0, -1, 0, 1 };
	int* xbuff = xbuff_ + i * _width * _height ;
	int* ybuff = ybuff_ + i * _width * _height ;
	//
	xbuff[0] = component.min_x;
	ybuff[0] = component.min_y;



	int x = xbuff[0];
	int y = ybuff[0];
	while (true)
	{
		x = xbuff[0];
		while (true)
		{
			if ((x >= 0 && x < _width) && (y >= 0 && y < _height) && (component.label == imgLabels[y * _width + x]))
			{
				xbuff[count] = x;
				ybuff[count] = y;
				

				++count;
			}
			if (x == component.max_x)
				break;
			x++;
		}

		if (y == component.max_y)
			break;
		y++;

	}
	//}





	int label = component.label;
	Pixel point;
	for (int c = 0; c < count; ++c)
	{
		int labels[numDirections] = { 0 };  // Left, right, top, bottom
		double minDiff = 1.0e10;
		int finalLabel = 0;
		labels[0] = imgLabels[(ybuff[c] * _width + _max(0, xbuff[c] - 1))];
		labels[1] = imgLabels[(ybuff[c] * _width + _min(_width - 1, xbuff[c] + 1))];
		labels[2] = imgLabels[(_max(0, ybuff[c] - 1) * _width + xbuff[c])];
		labels[3] = imgLabels[(_min(_height - 1, ybuff[c] + 1) * _width + xbuff[c])];
		/*	labels[4] = imgLabels.at<int>(max(0,ybuff[c] -1), max(0, xbuff[c] - 1));
		labels[5] = imgLabels.at<int>(max(0, ybuff[c] - 1), min(_width - 1, xbuff[c] + 1));
		labels[6] = imgLabels.at<int>(min(_height - 1, ybuff[c] + 1), max(0, xbuff[c] - 1));
		labels[7] = imgLabels.at<int>(min(_height - 1, ybuff[c] + 1), min(_width - 1, xbuff[c] + 1)); */
		/*	if (labels[0] == label && labels[2] == label && labels[1] == label && labels[3] == label)
		continue;*/
		if (labels[0] == labels[1] && labels[2] == labels[3] && labels[0] == labels[2])
			continue;
		//for (int i = 0; i < 4; i++)
		//{
		//	if (labels[i] != label)
		//	{
		//		sn = 1;
		//	}
		//}
		//if (sn == 0)
		//	continue;
		point.l = img[ybuff[c] * _width * 3 + xbuff[c] * 3 + 0];
		point.a = img[ybuff[c] * _width * 3 + xbuff[c] * 3 + 1];
		point.b = img[ybuff[c] * _width * 3 + xbuff[c] * 3 + 2];
		point.x = xbuff[c];
		point.y = ybuff[c];
		//minDiff = compute_distance(_components[label], point, _components[label].num, 1);
		for (int j = 0; j < 4; ++j)
		{
			double diff = 0;
			if (is_test)
				diff = compute_distance(_components[labels[j]], point, _components[labels[j]].num, 0);
			else
				diff = compute_distance(_components[labels[j]], point, _components[labels[j]].num, 1);

			if (diff < minDiff)
			{
				minDiff = diff;
				finalLabel = labels[j];
			}
		}
		//cout << minDiff << endl;
		if (finalLabel == label)
			continue;
		imgLabels[ybuff[c] * _width + xbuff[c]] = finalLabel;

		_components[finalLabel].lMean = (_components[finalLabel].lMean * _components[finalLabel].num + point.l) / (_components[finalLabel].num + 1);
		_components[finalLabel].aMean = (_components[finalLabel].aMean * _components[finalLabel].num + point.a) / (_components[finalLabel].num + 1);
		_components[finalLabel].bMean = (_components[finalLabel].bMean * _components[finalLabel].num + point.b) / (_components[finalLabel].num + 1);
		_components[finalLabel].xMean = (_components[finalLabel].xMean * _components[finalLabel].num + point.x) / (_components[finalLabel].num + 1);
		_components[finalLabel].yMean = (_components[finalLabel].yMean * _components[finalLabel].num + point.y) / (_components[finalLabel].num + 1);
		_components[finalLabel].num += 1;

		if (_components[label].num > 1)
		{
			_components[label].lMean = (_components[label].lMean * _components[label].num - point.l) / (_components[label].num - 1);
			_components[label].aMean = (_components[label].aMean * _components[label].num - point.a) / (_components[label].num - 1);
			_components[label].bMean = (_components[label].bMean * _components[label].num - point.b) / (_components[label].num - 1);
			_components[label].xMean = (_components[label].xMean * _components[label].num - point.x) / (_components[label].num - 1);
			_components[label].yMean = (_components[label].yMean * _components[label].num - point.y) / (_components[label].num - 1);
		}
		_components[label].num -= 1;


		//// Update start point
		//update_start_point(_components[finalLabel], point);

	}



	count = 1;
	xbuff[0] = component.max_x;
	ybuff[0] = component.max_y;




	x = xbuff[0];
	y = ybuff[0];
	while (true)
	{
		x = xbuff[0];
		while (true)
		{
			if ((x >= 0 && x < _width) && (y >= 0 && y < _height) && (component.label == imgLabels[(y * _width + x)]))
			{
				xbuff[count] = x;
				ybuff[count] = y;
			
				
				++count;
			}
			if (x == component.min_x)
				break;
			x--;
		}
		if (y == component.min_y)
			break;
		y--;
		//}

	}



	label = component.label;
	for (int c = 0; c < count; ++c)
	{
		int labels[numDirections] = { 0 };  // Left, right, top, bottom
		double minDiff = 1.0e10;
		int finalLabel = 0;
		labels[0] = imgLabels[(ybuff[c] * _width + _max(0, xbuff[c] - 1))];
		labels[1] = imgLabels[(ybuff[c] * _width + _min(_width - 1, xbuff[c] + 1))];
		labels[2] = imgLabels[(_max(0, ybuff[c] - 1) * _width + xbuff[c])];
		labels[3] = imgLabels[(_min(_height - 1, ybuff[c] + 1) * _width + xbuff[c])];
		if (labels[0] == labels[1] && labels[2] == labels[3] && labels[0] == labels[2])
			continue;


		point.l = img[ybuff[c] * _width * 3 + xbuff[c] * 3 + 0];
		point.a = img[ybuff[c] * _width * 3 + xbuff[c] * 3 + 1];
		point.b = img[ybuff[c] * _width * 3 + xbuff[c] * 3 + 2];
		point.x = xbuff[c];
		point.y = ybuff[c];

		for (int j = 0; j < 4; ++j)
		{
			double diff = compute_distance(_components[labels[j]], point, _components[labels[j]].num, 1);

			if (diff < minDiff)
			{
				minDiff = diff;
				finalLabel = labels[j];
			}
		}
		//cout << minDiff << endl;
		if (finalLabel == label)
			continue;

		imgLabels[(ybuff[c] * _width + xbuff[c])] = finalLabel;

	
		if (is_test)
		{
			_components[finalLabel].xMean = (_components[finalLabel].xMean * _components[finalLabel].num + point.x) / (_components[finalLabel].num + 1);
			_components[finalLabel].yMean = (_components[finalLabel].yMean * _components[finalLabel].num + point.y) / (_components[finalLabel].num + 1);
			_components[finalLabel].num += 1;

			if (_components[label].num > 1)
			{
				_components[label].xMean = (_components[label].xMean * _components[label].num - point.x) / (_components[label].num - 1);
				_components[label].yMean = (_components[label].yMean * _components[label].num - point.y) / (_components[label].num - 1);
			}
			_components[label].num -= 1;


			// Update start point
			/*update_start_point(_components[finalLabel], point);*/
		}
		else
		{
			_components[finalLabel].lMean = (_components[finalLabel].lMean * _components[finalLabel].num + point.l) / (_components[finalLabel].num + 1);
			_components[finalLabel].aMean = (_components[finalLabel].aMean * _components[finalLabel].num + point.a) / (_components[finalLabel].num + 1);
			_components[finalLabel].bMean = (_components[finalLabel].bMean * _components[finalLabel].num + point.b) / (_components[finalLabel].num + 1);
			_components[finalLabel].xMean = (_components[finalLabel].xMean * _components[finalLabel].num + point.x) / (_components[finalLabel].num + 1);
			_components[finalLabel].yMean = (_components[finalLabel].yMean * _components[finalLabel].num + point.y) / (_components[finalLabel].num + 1);
			_components[finalLabel].num += 1;

			if (_components[label].num > 1)
			{
				_components[label].lMean = (_components[label].lMean * _components[label].num - point.l) / (_components[label].num - 1);
				_components[label].aMean = (_components[label].aMean * _components[label].num - point.a) / (_components[label].num - 1);
				_components[label].bMean = (_components[label].bMean * _components[label].num - point.b) / (_components[label].num - 1);
				_components[label].xMean = (_components[label].xMean * _components[label].num - point.x) / (_components[label].num - 1);
				_components[label].yMean = (_components[label].yMean * _components[label].num - point.y) / (_components[label].num - 1);
			}
			_components[label].num -= 1;


		/* Update start point
		update_start_point(_components[finalLabel], point);*/
		}
	}
	////delete[] xbuff;
	////delete[] ybuff;
}


extern hipError_t clusterWithCuda(const Mat& img, Mat& imgLabels, Component* _components, const Score* scores, const bool is_test, const int _numsp)
{
	int _height = img.rows;
	int _width = img.cols;
	int* img_;
	int* imgLabels_;
	Component* _components_;
	Score* scores_;
	bool* is_test_;
	hipError_t status;
	int* _height_;
	int* _width_;
	int* xbuff_;
	int* ybuff_;
	status = hipSetDevice(0);

	if (status != hipSuccess)
	{
		cout << "hipSetDevice failed" << endl;
		
	}
	status = hipMalloc((void**)&img_, _height * _width * 3 * sizeof(int));
	if (status != hipSuccess)
	{
		cout << "hipMalloc failed1" << endl;
		//goto Error;
	}
	status = hipMalloc((void**)&imgLabels_, _height * _width * sizeof(int));
	if (status != hipSuccess)
	{
		cout << "hipMalloc failed2" << endl;
		//goto Error;
	}
	status = hipMalloc((void**)&_components_, _numsp * sizeof(Component));
	if (status != hipSuccess)
	{
		cout << "hipMalloc failed3" << endl;
		//goto Error;
	}
	status = hipMalloc((void**)&is_test_, sizeof(bool));
	if (status != hipSuccess)
	{
		cout << "hipMalloc failed4" << endl;
		//goto Error;
	}
	status = hipMalloc((void**)&scores_, _numsp * sizeof(Score));
	if (status != hipSuccess)
	{
		cout << "hipMalloc failed5" << endl;
		//goto Error;
	}
	status = hipMalloc((void**)&_height_, sizeof(int));
	if (status != hipSuccess)
	{
		cout << "hipMalloc failed6" << endl;
		//goto Error;
	}
	status = hipMalloc((void**)&_width_, sizeof(int));
	if (status != hipSuccess)
	{
		cout << "hipMalloc failed7" << endl;
		//goto Error;
	}
	status = hipMalloc((void**)&xbuff_, _numsp * _height * _width * sizeof(int));
	if (status != hipSuccess)
	{
		cout << "hipMalloc failed16" << endl;
		//goto Error;
	}
	status = hipMalloc((void**)&ybuff_, _numsp * _height * _width * sizeof(int));
	if (status != hipSuccess)
	{
		cout << "hipMalloc failed17" << endl;
		//goto Error;
	}
	int* img_tmp = new int[_width * _height * 3];
	for (int h = 0; h < _height; ++h)
	{
		for (int w = 0; w < _width; ++w)
		{
			img_tmp[h * _width * 3 + w * 3 + 0] = static_cast<int>(img.at<Vec3b>(h, w)[0]);
			img_tmp[h * _width * 3 + w * 3 + 1] = static_cast<int>(img.at<Vec3b>(h, w)[1]);
			img_tmp[h * _width * 3 + w * 3 + 2] = static_cast<int>(img.at<Vec3b>(h, w)[2]);
		}
	}
	status = hipMemcpy(img_, img_tmp, _width * _height * 3 * sizeof(int), hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		cout << "hipMemcpy failed8" << endl;
		//goto Error;
	}
	int* imgLabels_tmp = new int[_width * _height];
	for (int h = 0; h < _height; ++h)
	{
		for (int w = 0; w < _width; ++w)
		{
			imgLabels_tmp[h * _width + w] = imgLabels.at<int>(h, w);
		}
	}
	status = hipMemcpy(imgLabels_, imgLabels_tmp, _width * _height * sizeof(int), hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		cout << "hipMemcpy failed9" << endl;
		//goto Error;
	}
	status = hipMemcpy(_components_, _components, _numsp * sizeof(Component), hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		cout << "hipMemcpy failed10" << endl;
		//goto Error;
	}
	status = hipMemcpy(scores_, scores, _numsp * sizeof(Score), hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		cout << "hipMemcpy failed11" << endl;
		//goto Error;
	}
	status = hipMemcpy(is_test_, &is_test, sizeof(bool), hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		cout << "hipMemcpy failed12" << endl;
		//goto Error;
	}
	status = hipMemcpy(_width_, &_width, sizeof(int), hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		cout << "hipMemcpy failed13" << endl;
		//goto Error;
	}
	status = hipMemcpy(_height_, &_height, sizeof(int), hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		cout << "hipMemcpy failed14" << endl;
		//goto Error;
	}
	clustering << <1, _numsp >> >(img_, imgLabels_, _components_, scores_, is_test_, _height_, _width_, xbuff_, ybuff_);

	status = hipDeviceSynchronize();
	if (status != hipSuccess)
	{
		cout << "hipDeviceSynchronize failed" << endl;
		//goto Error;
	}






	status = hipMemcpy(imgLabels_tmp, imgLabels_, _width * _height * sizeof(int), hipMemcpyDeviceToHost);
	if (status != hipSuccess)
	{
		cout << "hipMemcpy failed15" << endl;
		//goto Error;
	}

	for (int h = 0; h < _height; ++h)
	{
		for (int w = 0; w < _width; ++w)
		{
			imgLabels.at<int>(h, w) = imgLabels_tmp[h * _width + w];
		}
	}


	hipFree(img_);
	hipFree(imgLabels_);
	hipFree(_components_);
	hipFree(scores_);
	hipFree(is_test_);
	hipFree(_height_);
	hipFree(_width_);
	hipFree(xbuff_);
	hipFree(ybuff_);
	delete[]img_tmp;
	delete[]imgLabels_tmp;
	return status;

}